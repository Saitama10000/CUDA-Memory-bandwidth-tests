#include "hip/hip_runtime.h"
/*
	I know I'm not checking for cuda errors but this is 
	just a small piece of code for a small benchmark.
*/

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void read(int* input, int* output, const long long size)
{
	long long gid = threadIdx.x + blockIdx.x * blockDim.x;
	if (gid > size) return;

	output[gid] = input[gid];
}


int main()
{
	const long long size = 1ll << 28;
	int* d_input;
	int* d_output;
	hipMalloc((void**)&d_input, size * sizeof(int));
	hipMalloc((void**)&d_output, size * sizeof(int));

	int blockSize = 256;
	int gridSize = size / blockSize;
	read << <gridSize, blockSize >> > (d_input, d_output, size);

	hipFree(d_input);
	hipFree(d_output);
	return 0;
}