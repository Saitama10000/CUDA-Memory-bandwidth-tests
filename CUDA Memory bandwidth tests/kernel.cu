#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "error.h"
#include "kernel.ptx"
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdio.h>

int main()
{
	hipDevice_t device;
	hipCtx_t context;
	hipModule_t module;
	hipFunction_t get_clock;
	hipFunction_t get_time;

	cuErr(hipInit(0));
	cuErr(hipDeviceGet(&device, 0));
	cuErr(hipCtxCreate(&context, 0, device));
	cuErr(hipModuleLoadData(&module, kernel_ptx));
	cuErr(hipModuleGetFunction(&get_clock, module, "get_clock"));
	cuErr(hipModuleGetFunction(&get_time, module, "get_time"));

	unsigned long long h_clock[1];
	unsigned long long clock = 0;
	unsigned long long time = 0;
	unsigned long long n = 0;
	
	hipDeviceptr_t d_clock;
	cuErr(hipMalloc(&d_clock, sizeof(h_clock[0])));
	
	while(true)
	{
		{
			void* args[] = { &d_clock };
			cuErr(hipModuleLaunchKernel(get_clock, 1, 1, 1, 1, 1, 1, 0, NULL, (void**)args, NULL));
			cuErr(hipMemcpyDtoH((void*)&h_clock, d_clock, sizeof(h_clock[0])));
			clock += h_clock[0];
		}
		{
			void* args[] = { &d_clock };
			cuErr(hipModuleLaunchKernel(get_time, 1, 1, 1, 1, 1, 1, 0, NULL, (void**)args, NULL));
			cuErr(hipMemcpyDtoH((void*)&h_clock, d_clock, sizeof(h_clock[0])));
			time += h_clock[0];
		}
		n += 1;
		printf("\rClock: %8.2f Time: %8.2f", (double)(clock) / n, (double)(time) / n);
	}

	cudaErr(hipDeviceReset());
	return 0;
}