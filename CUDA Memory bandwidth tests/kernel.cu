#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "error.h"
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdio.h>

extern "C" __global__ void getClock(int* timings)
{
	int tid = threadIdx.x;
	unsigned long long start = clock64();

	unsigned long long end = clock64();
	timings[tid] = start;
	timings[tid + 1] = end;
}

int main()
{
	const long long size = 1ll << 28;
	int* d_input;
	int* d_output;
	cudaErr(hipMalloc((void**)&d_input, size * sizeof(int)));
	cudaErr(hipMalloc((void**)&d_output, size * sizeof(int)));


	hipDevice_t device;
	hipCtx_t context;
	hipModule_t module;
	hipFunction_t get_clock;
	hipFunction_t get_time;

	cuErr(hipInit(0));
	cuErr(hipDeviceGet(&device, 0));
	cuErr(hipCtxCreate(&context, 0, device));
	cuErr(hipModuleLoad(&module, "kernel.ptx"));
	cuErr(hipModuleGetFunction(&get_clock, module, "get_clock"));
	cuErr(hipModuleGetFunction(&get_time, module, "get_time"));

	unsigned long long h_clock[2];
	hipDeviceptr_t d_clock;
	cuErr(hipMalloc(&d_clock, 2 * sizeof(h_clock[0])));

	for (int i = 0; i < 16; i++)
	{
		{
			void* args[] = { &d_clock };
			cuErr(hipModuleLaunchKernel(get_clock, 1, 1, 1, 1, 1, 1, 0, NULL, (void**)args, NULL));
			cuErr(hipMemcpyDtoH((void*)&h_clock, d_clock, 2 * sizeof(h_clock[0])));
			printf("Clocks: %lld %lld %lld\n", h_clock[0], h_clock[1], h_clock[1] - h_clock[0]);
		}
		{
			void* args[] = { &d_clock };
			cuErr(hipModuleLaunchKernel(get_time, 1, 1, 1, 1, 1, 1, 0, NULL, (void**)args, NULL));
			cuErr(hipMemcpyDtoH((void*)&h_clock, d_clock, 2 * sizeof(h_clock[0])));
			printf("Time: %lld %lld %lld\n", h_clock[0], h_clock[1], h_clock[1] - h_clock[0]);
		}
		{
			getClock<<<1, 1>>>((void*)d_clock);
			cudaErr(hipMemcpy((void*)&h_clock, (void*)d_clock, 2 * sizeof(h_clock[0]), hipMemcpyDeviceToHost));
			printf("Time: %lld %lld %lld\n", h_clock[0], h_clock[1], h_clock[1] - h_clock[0]);
		}
	}



	cudaErr(hipFree(d_input));
	cudaErr(hipFree(d_output));
	hipDeviceReset();
	return 0;
}